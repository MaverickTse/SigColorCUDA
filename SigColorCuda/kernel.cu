#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include "filter.h"
#include <string>
#include <sstream>
#include <fstream>

template <typename T>
__device__ void cuYC48_RGB48(T* px)
{
	float y = static_cast<float>(px[0]);
	float cb = static_cast<float>(px[1]);
	float cr = static_cast<float>(px[2]);
	float r = 1.0037736040867458f*y + 0.0009812686948862392f*cb + 1.4028706125758748f*cr;
	float g = 1.0031713814217937f*y + -0.34182057237626395f*cb + -0.7126004638855613f*cr;
	float b = 1.0038646965904563f*y + 1.7738420513779833f*cb + 0.0018494308641594699f*cr;
	px[0] = static_cast<T>(r);
	px[1] = static_cast<T>(g);
	px[2] = static_cast<T>(b);
}

template <typename T>
__device__ void cuRGB48_YC48(T* px)
{
	float r = static_cast<float>(px[0]);
	float g = static_cast<float>(px[1]);
	float b = static_cast<float>(px[2]);
	float y = 0.297607421875f*r + 0.586181640625f*g + 0.11279296875f*b;
	float cb = -0.1689453125f*r + -0.331298828125f*g + 0.5f*b;
	float cr = 0.5f*r + -0.419189453125f*g + -0.0810546875f*b;
	px[0] = static_cast<T>(y);
	px[1] = static_cast<T>(cb);
	px[2] = static_cast<T>(cr);
}

template <typename T>
__device__ void cuSig(T* px, float midtone, float strength)
{
	float i = static_cast<float>(*px) / 4096.0f;
	float a = midtone;
	float b = strength;
	float term_a, term_b, term_c;
	term_a = 1.0f / (1.0f + expf(b*(a - i)));
	term_b = 1.0f / (1.0f + expf(b*(a - 1.0f)));
	term_c = 1.0f / (1.0f + expf(a*b));
	float s = ((term_a - term_c) / (term_b - term_c))*4096.0;
	*px = static_cast<T>(s);
}

template <typename T>
__device__ void cuLogit(T* px, float midtone, float strength)
{
	float i = static_cast<float>(*px) / 4096.0f;
	float a = midtone;
	float b = strength;
	float term_b, term_c;
	term_b = 1.0f / (1.0f + expf(b*(a - 1.0f)));
	term_c = 1.0f / (1.0f + expf(a*b));
	float l = a - (logf(1.0f / (i*(term_b - term_c) + term_c) - 1.0f) / b);

	*px = static_cast<T>(l*4096.0f);
}

__global__ void SigmodialTransform(short* image, size_t width, size_t height, size_t stride, float midtone, float strength, bool r = true, bool g = true, bool b = true)
{
	auto x = blockIdx.x * blockDim.x + threadIdx.x;
	auto y = blockIdx.y* blockDim.y + threadIdx.y;
	if ((x > width) || (y > height)) return;
	short* px = reinterpret_cast<short*>(reinterpret_cast<unsigned char*>(image) + stride*y + x * sizeof(short) * 3);
	float fpx[3] = { static_cast<float>(px[0]), static_cast<float>(px[1]), static_cast<float>(px[2]) };
	cuYC48_RGB48(fpx);
	if (r) cuSig(fpx, midtone, strength);
	if (g) cuSig(fpx+1, midtone, strength);
	if (b) cuSig(fpx+2, midtone, strength);
	cuRGB48_YC48(fpx);
	px[0] = static_cast<short>(fpx[0]);
	px[1] = static_cast<short>(fpx[1]);
	px[2] = static_cast<short>(fpx[2]);
}

__global__ void LogitTransform(short* image, size_t width, size_t height, size_t stride, float midtone, float strength, bool r = true, bool g = true, bool b = true)
{
	auto x = blockIdx.x * blockDim.x + threadIdx.x;
	auto y = blockIdx.y* blockDim.y + threadIdx.y;
	if ((x > width) || (y > height)) return;
	short* px = reinterpret_cast<short*>(reinterpret_cast<unsigned char*>(image) + stride*y + x * sizeof(short) * 3);
	float fpx[3] = { static_cast<float>(px[0]), static_cast<float>(px[1]), static_cast<float>(px[2]) };
	cuYC48_RGB48(fpx);
	if (r) cuLogit(fpx, midtone, strength);
	if (g) cuLogit(fpx+1, midtone, strength);
	if (b) cuLogit(fpx+2, midtone, strength);
	cuRGB48_YC48(fpx);
	px[0] = static_cast<short>(fpx[0]);
	px[1] = static_cast<short>(fpx[1]);
	px[2] = static_cast<short>(fpx[2]);
}


hipError_t utlSLTransform(void* ycp_edit, int w, int h, int max_w, void* dev_buffer, size_t dev_stride, float midtone, float strength, bool mode = true, bool r=true, bool g=true, bool b=true)
{
	hipError_t err = hipMemcpy2D(dev_buffer, dev_stride, ycp_edit, max_w * sizeof(short) * 3, w * sizeof(short) * 3, h, hipMemcpyHostToDevice);
	if (err != hipSuccess) return err;
	dim3 threadsperblock(32, 32);
	dim3 numBlocks((w / threadsperblock.x) + 1, (h / threadsperblock.y) + 1);

	if (mode)
	{
		SigmodialTransform <<<numBlocks, threadsperblock>>> (reinterpret_cast<short*>(dev_buffer), w, h, dev_stride, midtone, strength, r, g, b);
	}
	else
	{
		LogitTransform <<<numBlocks, threadsperblock >>> (reinterpret_cast<short*>(dev_buffer), w, h, dev_stride, midtone, strength, r, g, b);
	}
	err = hipGetLastError();
	if (err != hipSuccess) return err;
	err = hipMemcpy2D(ycp_edit, max_w * 3 * sizeof(short), dev_buffer, dev_stride, w * 3 * sizeof(short), h, hipMemcpyDeviceToHost);
	return err;
}

#ifdef _DEBUG
#define PLUGIN_NAME_S "SContrast CUDA (DEBUG)"
#define PLUGIN_NAME_D "SDeContrast CUDA (DEBUG)"
#define VERSION_S "SContrast CUDA (DEBUG) v0.01 by MT"
#define VERSION_D "SDeContrast CUDA (DEBUG) v0.01 by MT"
#else
#define PLUGIN_NAME_S "SContrast CUDA"
#define PLUGIN_NAME_D "SDeContrast CUDA"
#define VERSION_S "SContrast CUDA v0.01 by MT"
#define VERSION_D "SDeContrast CUDA v0.01 by MT"
#endif

std::ofstream logfilec("sigcontrast_log.csv", std::ios_base::app);
std::ofstream logfiled("sigdecontrast_log.csv", std::ios_base::app);
std::ostringstream logcbuf;
std::ostringstream logdbuf;
char* slider_label[] = { "Midtone", "Strength" };
int slider_default[] = { 50, 5 };
int slider_min[] = { 1, 1 };
int slider_max[] = { 100, 20 };
int trackN = 2;

char* box_label[] = { "R", "G", "B", "Show Benchmark", "Log Benchmark when saving" };
int box_default[] = { 1,1,1,0,0 };
int checkN = 5;

hipError_t errors = hipSuccess;
hipError_t errord = hipSuccess;
hipEvent_t start, stop, startd, stopd;
float msc, msd;
void* dev_SC = nullptr;
void* dev_DC = nullptr;
size_t strides = 0;
size_t strided = 0;

FILTER_DLL SC = {
	FILTER_FLAG_EX_INFORMATION | FILTER_FLAG_PRIORITY_LOWEST,	//	filter flags, use bitwise OR to add more
	0, 0,						//	dialogbox size
	PLUGIN_NAME_S,			//	Filter plugin name
	trackN,					//	トラックバーの数 (0なら名前初期値等もNULLでよい)
	slider_label ,					//	slider label names in English
	slider_default,				//	トラックバーの初期値郡へのポインタ
	slider_min, slider_max,			//	トラックバーの数値の下限上限 (NULLなら全て0～256)
	checkN,					//	チェックボックスの数 (0なら名前初期値等もNULLでよい)
	box_label,					//	チェックボックスの名前郡へのポインタ
	box_default,				//	チェックボックスの初期値郡へのポインタ
	func_proc_s,					//	main filter function, use NULL to skip
	func_init_s,						//	initialization function, use NULL to skip
	func_exit_s,						//	on-exit function, use NULL to skip
	func_update_s,						//	invokes when when settings changed. use NULL to skip
	func_WndProc_s,						//	for capturing dialog's control messages. Essential if you use button or auto uncheck some checkboxes.
	NULL, NULL,					//	Reserved. Do not use.
	NULL,						//  pointer to extra data when FILTER_FLAG_EX_DATA is set
	NULL,						//  extra data size
	VERSION_S,
	//  pointer or c-string for full filter info when FILTER_FLAG_EX_INFORMATION is set.
	NULL,						//	invoke just before saving starts. NULL to skip
	func_save_end_s,						//	invoke just after saving ends. NULL to skip
};

FILTER_DLL SD = {
	FILTER_FLAG_EX_INFORMATION | FILTER_FLAG_PRIORITY_LOWEST,	//	filter flags, use bitwise OR to add more
	0, 0,						//	dialogbox size
	PLUGIN_NAME_D,			//	Filter plugin name
	trackN,					//	トラックバーの数 (0なら名前初期値等もNULLでよい)
	slider_label ,					//	slider label names in English
	slider_default,				//	トラックバーの初期値郡へのポインタ
	slider_min, slider_max,			//	トラックバーの数値の下限上限 (NULLなら全て0～256)
	checkN,					//	チェックボックスの数 (0なら名前初期値等もNULLでよい)
	box_label,					//	チェックボックスの名前郡へのポインタ
	box_default,				//	チェックボックスの初期値郡へのポインタ
	func_proc_d,					//	main filter function, use NULL to skip
	func_init_d,						//	initialization function, use NULL to skip
	func_exit_d,						//	on-exit function, use NULL to skip
	func_update_d,						//	invokes when when settings changed. use NULL to skip
	func_WndProc_d,						//	for capturing dialog's control messages. Essential if you use button or auto uncheck some checkboxes.
	NULL, NULL,					//	Reserved. Do not use.
	NULL,						//  pointer to extra data when FILTER_FLAG_EX_DATA is set
	NULL,						//  extra data size
	VERSION_D,
	//  pointer or c-string for full filter info when FILTER_FLAG_EX_INFORMATION is set.
	NULL,						//	invoke just before saving starts. NULL to skip
	func_save_end_d,						//	invoke just after saving ends. NULL to skip
};


FILTER_DLL* pluginlist[] = { &SC, &SD, nullptr };
// Export the above filter table
EXTERN_C  __declspec(dllexport) FILTER_DLL** GetFilterTableList(void)
{

	return pluginlist;
}

BOOL func_init_s(FILTER *fp)
{
	errors = hipSetDevice(0);
	if (errors != hipSuccess)
	{
		MessageBox(fp->hwnd, "Cannot set default CUDA device!", "SContrast CUDA Error", MB_OK | MB_ICONERROR);
		return FALSE;
	}
	hipEventCreate(&start);
	hipEventCreate(&stop);
	return TRUE;
}

BOOL func_proc_s(FILTER *fp, FILTER_PROC_INFO *fpip)
{
	hipEventRecord(start);
	if (!dev_SC)
	{
		errors = hipMallocPitch(&dev_SC, &strides, fpip->max_w*3*sizeof(short), fpip->max_h);
		if (errors != hipSuccess)
		{
			MessageBox(fp->hwnd, "Not enough Video Memory!", "SContrast CUDA Error", MB_OK | MB_ICONERROR);
			return FALSE;
		}
	}
	float midtone = static_cast<float>(fp->track[0]) / 100.0f;
	float strength = static_cast<float>(fp->track[1]);
	errors = utlSLTransform((void*)fpip->ycp_edit, fpip->w, fpip->h, fpip->max_w, dev_SC, strides, midtone, strength, true, (fp->check[0]==1), (fp->check[1]==1), (fp->check[2]==1));
	if (errors != hipSuccess)
	{
		MessageBox(fp->hwnd, "Kernel Error!", "SContrast CUDA Error", MB_OK | MB_ICONERROR);
		MessageBox(fp->hwnd, hipGetErrorString(errors), "SContrast CUDA_Error", MB_OK);
		return FALSE;
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&msc, start, stop);
	
	if (fp->check[3] && !(fp->exfunc->is_saving(fpip->editp)))
	{
		std::string msg = "SCon: " + std::to_string(msc) + "ms @" + std::to_string(fpip->w) + "x" + std::to_string(fpip->h);
		SetWindowText(fp->hwnd, msg.c_str());
	}

	if (fp->check[4] && fp->exfunc->is_saving(fpip->editp))
	{
		// write into buffer
		logcbuf << std::to_string(fpip->frame) << ", " << std::to_string(fpip->w) << ", " << std::to_string(fpip->h) << ", " << std::to_string(msc) << std::endl;
		if (logcbuf.tellp() >= 65536)
		{
			logfilec << logcbuf.str();
			logfilec.flush();
			logcbuf.str(std::string());
		}
	}
	
	return TRUE;
}

BOOL func_exit_s(FILTER *fp)
{
	logfilec.flush();
	logcbuf.flush();
	logfilec.close();
	//hipEventDestroy(start);
	//hipEventDestroy(stop);
	//if (dev_SC) errors = hipFree(dev_SC);
	//if (errors != hipSuccess)
	//{
	//	MessageBox(fp->hwnd, "hipFree error on Exit!", "SContrast CUDA Error", MB_OK | MB_ICONERROR);
	//	return FALSE;
	//}
	errors = hipDeviceReset();
	if (errors != hipSuccess)
	{
		MessageBox(fp->hwnd, "Device cleanup error!", "SContrast CUDA Error", MB_OK | MB_ICONERROR);
		return FALSE;
	}
	return TRUE;
}

BOOL func_save_end_s(FILTER *fp, void *editp)
{
	if (fp->check[4])
	{
		logfilec << logcbuf.str();
		logfilec.flush();
		logcbuf.str(std::string());
	}
	return TRUE;
}

BOOL func_update_s(FILTER *fp, int status)
{
	switch (status)
	{
		case FILTER_UPDATE_STATUS_CHECK + 3:
		{
			if (fp->check[3] == 0)
			{
				SetWindowText(fp->hwnd, PLUGIN_NAME_S);
				return FALSE;
			}
		}break;

	}
	return TRUE;
}

BOOL func_WndProc_s(HWND hwnd, UINT message, WPARAM wparam, LPARAM lparam, void *editp, FILTER *fp)
{
	switch (message)
	{
	case WM_FILTER_FILE_CLOSE:
	{
		if (dev_SC)
		{
			hipFree(dev_SC);
			dev_SC = nullptr;
		};
		break;
	}
	};
	return FALSE;
}

/****************************************************************************************/
/****************************************************************************************/
/****************************************************************************************/
/****************************************************************************************/
/***************** DeContrast Plugin ****************************************************/
/****************************************************************************************/

BOOL func_init_d(FILTER *fp)
{
	errord = hipSetDevice(0);
	if (errord != hipSuccess)
	{
		MessageBox(fp->hwnd, "Cannot set default CUDA device!", "SDeContrast CUDA Error", MB_OK | MB_ICONERROR);
		return FALSE;
	}
	hipEventCreate(&startd);
	hipEventCreate(&stopd);
	return TRUE;
}

BOOL func_proc_d(FILTER *fp, FILTER_PROC_INFO *fpip)
{
	hipEventRecord(startd);
	if (!dev_DC)
	{
		errord = hipMallocPitch(&dev_DC, &strided, fpip->max_w * 3 * sizeof(short), fpip->max_h);
		if (errord != hipSuccess)
		{
			MessageBox(fp->hwnd, "Not enough Video Memory!", "SDeContrast CUDA Error", MB_OK | MB_ICONERROR);
			return FALSE;
		}
	}
	float midtone = static_cast<float>(fp->track[0]) / 100.0f;
	float strength = static_cast<float>(fp->track[1]);
	errord = utlSLTransform((void*)fpip->ycp_edit, fpip->w, fpip->h, fpip->max_w, dev_DC, strided, midtone, strength, false, (fp->check[0] == 1), (fp->check[1] == 1), (fp->check[2] == 1));
	if (errord != hipSuccess)
	{
		MessageBox(fp->hwnd, "Kernel Error!", "SDeContrast CUDA Error", MB_OK | MB_ICONERROR);
		MessageBox(fp->hwnd, hipGetErrorString(errord), "SDeContrast CUDA_Error", MB_OK);
		return FALSE;
	}
	hipEventRecord(stopd);
	hipEventSynchronize(stopd);
	hipEventElapsedTime(&msd, startd, stopd);

	if (fp->check[3] && !(fp->exfunc->is_saving(fpip->editp)))
	{
		std::string msg = "SDeCon: " + std::to_string(msd) + "ms @" + std::to_string(fpip->w) + "x" + std::to_string(fpip->h);
		SetWindowText(fp->hwnd, msg.c_str());
	}

	if (fp->check[4] && fp->exfunc->is_saving(fpip->editp))
	{
		// write into buffer
		logdbuf << std::to_string(fpip->frame) << ", " << std::to_string(fpip->w) << ", " << std::to_string(fpip->h) << ", " << std::to_string(msd) << std::endl;
		if (logdbuf.tellp() >= 65536)
		{
			logfiled << logdbuf.str();
			logfiled.flush();
			logdbuf.str(std::string());
		}
	}

	return TRUE;
}

BOOL func_exit_d(FILTER *fp)
{
	logfiled.flush();
	logdbuf.flush();
	logfiled.close();
	//hipEventDestroy(startd);
	//hipEventDestroy(stopd);
	//if (dev_DC) errord = hipFree(dev_DC);
	//if (errord != hipSuccess)
	//{
	//	MessageBox(fp->hwnd, "hipFree error on Exit!", "SDeContrast CUDA Error", MB_OK | MB_ICONERROR);
	//	return FALSE;
	//}
	errord = hipDeviceReset();
	if (errord != hipSuccess)
	{
		MessageBox(fp->hwnd, "Device cleanup error!", "SDeContrast CUDA Error", MB_OK | MB_ICONERROR);
		return FALSE;
	}
	return TRUE;
}

BOOL func_save_end_d(FILTER *fp, void *editp)
{
	if (fp->check[4])
	{
		logfiled << logdbuf.str();
		logfiled.flush();
		logdbuf.str(std::string());
	}
	return TRUE;
}

BOOL func_update_d(FILTER *fp, int status)
{
	switch (status)
	{
	case FILTER_UPDATE_STATUS_CHECK + 3:
	{
		if (fp->check[3] == 0)
		{
			SetWindowText(fp->hwnd, PLUGIN_NAME_D);
			return FALSE;
		}
	}break;

	}
	return TRUE;
}

BOOL func_WndProc_d(HWND hwnd, UINT message, WPARAM wparam, LPARAM lparam, void *editp, FILTER *fp)
{
	switch (message)
	{
	case WM_FILTER_FILE_CLOSE:
	{
		if (dev_DC)
		{
			hipFree(dev_DC);
			dev_DC = nullptr;
		};
		break;
	}
	};
	return FALSE;
}